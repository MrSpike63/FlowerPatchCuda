#include "hip/hip_runtime.h"
// nvcc -o flower flower.cu -O3 -m=64 -Xptxas -O3
//nvcc flower.cu -o flower.exe -O3 -m=64 -Xptxas -v -Iboinc\ -Iboinc\win\ -Lboinc\lib\win\ -lboinc_api -lboinc -lcuda -luser32 -DBOINC -D_WIN32
#include <iostream>

#define LCG_XOR 25214903917L
#define LCG_MULTIPLIER 25214903917L
#define LCG_ADDEND 11L
#define LCG_MASK (1ULL << 48) - 1


#define TOTAL_WORK (1ULL << 38)             // How many seeds to check (per task). (Each 2^48 seedspace has 6 offsets, so each offset
                                            // has a seedspace of (2^48 / 6)
#define THREAD_WORK (1ULL << 22)            // Seeds that each thread has to check
#define THREADS_PER_BLOCK 128               // Threads per kernel block
#define BLOCKS_PER_GROUP 64                 // How many blocks to do in each group (checkpoint occurs after each block group)

#define FLOWER_COUNT 18                     // Total flowers that are being matched
#define RETURN_THRESHOLD 2                  // Threshold for missing_count. missing_count < RETURN_THRESHOLD

#define DEBUG true


// Calculated with lcg_skip_calculator.cpp (calculating 2^35 lcg skips is slow)
#define LCG_SKIP_THREAD_WORK_MULTIPLIER 43720451817473ULL
#define LCG_SKIP_THREAD_WORK_ADDEND 186612394754048ULL

#define LCG_SKIP_GROUP_MULTIPLIER 121770912776193ULL
#define LCG_SKIP_GROUP_ADDEND 38139309588480ULL

#define LCG_SKIP_2_16_MULTIPLIER 184682916610049ULL
#define LCG_SKIP_2_16_ADDEND 258238410457088ULL

#define LCG_SKIP_2_32_MULTIPLIER 237099374608385ULL
#define LCG_SKIP_2_32_ADDEND 229492987527168ULL


// #define BOINC 0
#ifdef BOINC
    #include "boinc_api.h"
    #if defined _WIN32 || defined _WIN64
        #include "boinc_win.h"
    #endif
#endif

#ifndef BOINC
    #define boinc_begin_critical_section()
    #define boinc_end_critical_section()
    #define boinc_finish(status)
    #define boinc_fraction_done(fraction)
    #define boinc_checkpoint_completed()
    #define boinc_fopen(arg1, arg2) fopen(arg1, arg2)
    #define boinc_delete_file(arg1) remove(arg1)
#endif

__constant__ bool known[15][7][15];

int center[3] = {99, 68, -100};
int coords[18][3] = {
    {97, 68, -100},
    {99, 68, -102},
    {100, 68, -100}, 
    {100, 68, -99},
    {98, 68, -98},
    {100, 69, -103},
    {95, 68, -102},
    {99, 68, -98},
    {97, 69, -104},
    {96, 67, -98},
    {94, 68, -100},
    {99, 68, -97},
    {102, 68, -102},
    {100, 68, -96},
    {93, 67, -99},
    {102, 67, -96},
    {95, 70, -107},
    {105, 68, -101}
};


// Polymetrics test data
/*
int center[3] = {-715, 79, -130};
int coords[18][3] = {
    {-716,     79,   -126},
    {-713,     79,   -132},
    {-713,     79,   -129},
    {-720,     79,   -128},
    {-718,     79,   -132},
    {-712,     79,   -131},
    {-715,     79,   -128},
    {-712,     78,   -134},
    {-709,     78,   -128},
    {-717,     79,   -132},
    {-709,     78,   -131},
    {-716,     78,   -128},
    {-718,     79,   -129},
    {-713,     79,   -128},
    {-717,     79,   -130},
    {-719,     79,   -130},
    {-714,     79,   -131},
    {-715,     79,   -127},
};
*/




__device__ __forceinline__ unsigned long long next(unsigned long long *seed, short bits) {
    *seed = (*seed * LCG_MULTIPLIER + LCG_ADDEND) & LCG_MASK;
    return *seed >> (48 - bits);
}

__device__ __forceinline__ unsigned int next_int(unsigned long long *seed, short bound) {
    
    if ((bound & -bound) == bound) {
        *seed = (*seed * LCG_MULTIPLIER + LCG_ADDEND) & LCG_MASK;
        return (int)((bound * (*seed >> 17)) >> 31);
    }
    
    int bits, value;
    do {
        *seed = (*seed * LCG_MULTIPLIER + LCG_ADDEND) & LCG_MASK;
        bits =  *seed >> 17;
        value = bits % bound;
    } while (bits - value + (bound - 1) < 0);

    return value;
}


__global__ void gpu_bruteforce(unsigned long long start_seed, unsigned long long *counter, unsigned long long *seed_list, int8_t *missing_count_list, unsigned long long *lcg_skip_list_multiplier, unsigned long long *lcg_skip_list_addend) {
    // Give each thread a unique identifier (to calculate dfz offsets)
    int thread_number = blockIdx.x * blockDim.x + threadIdx.x;

    unsigned long long seed = (start_seed * lcg_skip_list_multiplier[thread_number] + lcg_skip_list_addend[thread_number]) & LCG_MASK;
    // seed = seed ^ LCG_XOR;

    int8_t missing_count = FLOWER_COUNT;

    int8_t fcount[15][7][15];
    __shared__ int8_t i_buffer[64][THREADS_PER_BLOCK];
    __shared__ int8_t j_buffer[64][THREADS_PER_BLOCK];
    __shared__ int8_t k_buffer[64][THREADS_PER_BLOCK];
    
    // Initialize circular buffer
    for (int8_t i = 0; i < 64; i++) {

        // Probably not needed, but ust to be safe.
        int8_t ptr = (int8_t) (i & 63); // int ptr = i;

        int8_t i_1 = (7 + next_int(&seed, 8)) - next_int(&seed, 8);
        int8_t j_1 = (3 + next_int(&seed, 4)) - next_int(&seed, 4);
        int8_t k_1 = (7 + next_int(&seed, 8)) - next_int(&seed, 8);
        i_buffer[ptr][threadIdx.x] = i_1;
        j_buffer[ptr][threadIdx.x] = j_1;
        k_buffer[ptr][threadIdx.x] = k_1;

        if (known[i_1][j_1][k_1] && fcount[i_1][j_1][k_1] == 0) {
            missing_count--;
        }


        fcount[i_1][j_1][k_1]++;
    }

    // Do work assigned to thread
    for (long time = 0; time < THREAD_WORK; time++) {

        // Check if current flowers are a perfect match
        if (missing_count < RETURN_THRESHOLD) {
            // Find how many dfz skips were made at kernel launch
            unsigned long long dfz = (time * 6) + (thread_number * THREAD_WORK * 6);

            // Safely store in array
            atomicAdd(counter, 1);
            seed_list[*counter - 1] = dfz; // Let cpu figure the rest out lmao
            missing_count_list[*counter - 1] = missing_count;
        }

        int8_t ptr = (int8_t) (time & 63);

        // Remove previous flower
        int8_t i_0 = i_buffer[ptr][threadIdx.x];
        int8_t j_0 = j_buffer[ptr][threadIdx.x];
        int8_t k_0 = k_buffer[ptr][threadIdx.x];

        fcount[i_0][j_0][k_0]--;

        if (known[i_0][j_0][k_0] && fcount[i_0][j_0][k_0] == 0) {
            missing_count++;
        }
        
        // Add new flower
        int8_t i_1 = (7 + next_int(&seed, 8)) - next_int(&seed, 8);
        int8_t j_1 = (3 + next_int(&seed, 4)) - next_int(&seed, 4);
        int8_t k_1 = (7 + next_int(&seed, 8)) - next_int(&seed, 8); 

        i_buffer[ptr][threadIdx.x] = i_1;
        j_buffer[ptr][threadIdx.x] = j_1;
        k_buffer[ptr][threadIdx.x] = k_1;

        if (known[i_1][j_1][k_1] && fcount[i_1][j_1][k_1] == 0) {
            missing_count--;
        }

        fcount[i_1][j_1][k_1]++;
    }
}



struct checkpoint_vars {
    int offset;
    unsigned long long dfz_tracker;
    unsigned long long seed;
    time_t elapsed_chkpoint;
    unsigned long long total_counter;
};

void run_kernel(unsigned long long dfz_initial_start) {

    // Counter of seeds found per block
    unsigned long long *counter = 0;
    hipMallocManaged(&counter, sizeof(unsigned long long));

    // List of seeds
    unsigned long long *seed_list;
    hipMallocManaged(&seed_list, 100 * sizeof(unsigned long long));

    // The missing_list for each seed found
    int8_t *missing_count_list;
    hipMallocManaged(&missing_count_list, 100 * sizeof(int8_t));

    // Total counter (total seeds found)
    unsigned long long total_counter = 0;




    // Calculate relative coordinates
    bool known2[15][7][15] = {false};
    for (short i = 0; i < FLOWER_COUNT; i++) {
        known2[coords[i][0] - center[0] + 7][coords[i][1] - center[1] + 3][coords[i][2] - center[2] + 7] = true;
    }

    // Pre calculate lcg skips
    unsigned long long *lcg_skip_list_multiplier;
    unsigned long long *lcg_skip_list_addend;
    hipMallocManaged(&lcg_skip_list_multiplier, THREADS_PER_BLOCK * BLOCKS_PER_GROUP * sizeof(unsigned long long));
    hipMallocManaged(&lcg_skip_list_addend, THREADS_PER_BLOCK * BLOCKS_PER_GROUP * sizeof(unsigned long long));
    
    // Calculate skip lists for the first kernel launch
    unsigned long long multiplier = 1;
    unsigned long long addend = 0;

    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS_PER_GROUP; i++) {
        lcg_skip_list_multiplier[i] = multiplier;
        lcg_skip_list_addend[i] = addend;

        addend = (addend * LCG_SKIP_THREAD_WORK_MULTIPLIER + LCG_SKIP_THREAD_WORK_ADDEND) & LCG_MASK;
        multiplier = (multiplier * LCG_SKIP_THREAD_WORK_MULTIPLIER) & LCG_MASK;
    }


    // Copy to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(known), &known2, sizeof(known2));


    // Total threads needed: (total seeds to check / the seeds per thread)
    int threads_needed = TOTAL_WORK / THREAD_WORK;

    // Total blocks needed: (total threads needed / threads per block) (round up to be safe)
    int total_blocks = (threads_needed + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;


    // Block group to start at
    int block_group_start = 0;
    time_t elapsed_chkpoint = 0;

    // Variables to track progress
    unsigned long long seed = 0;
    unsigned long long dfz_tracker = dfz_initial_start;

    FILE *fp;

    boinc_begin_critical_section();

    // Load checkpoint (if any)
    fp = boinc_fopen("flower_checkpoint.txt", "r");
    if (fp) {
        struct checkpoint_vars data_store;
        fread(&data_store, sizeof(data_store), 1, fp);

        block_group_start = data_store.offset;
        dfz_tracker = data_store.dfz_tracker;
        seed = data_store.seed;
        elapsed_chkpoint = data_store.elapsed_chkpoint;
        total_counter = data_store.total_counter;

        // Linux :c
        // fprintf(stderr, "Checkpoint loaded. Offset: %d, task time: %lu s.\n", block_group_start, elapsed_chkpoint);
        fprintf(stderr, "Checkpoint loaded. Offset: %d, task time: %llu s.\n", block_group_start, elapsed_chkpoint);
        fclose(fp);
    } else {
        //
        // Calculate the seed from the starting dfz given as an argument
        //

        // Large skips (2^32)
        int large_skips = floor((double)dfz_initial_start / (1ULL << 32));
        for (int i = 0; i < large_skips; i++) {
            seed = (seed * LCG_SKIP_2_32_MULTIPLIER + LCG_SKIP_2_32_ADDEND) & LCG_MASK;
        }
        dfz_initial_start -= large_skips * (1ULL << 32);

        // Medium skips (2^16)
        int medium_skips = floor((double)dfz_initial_start / (1ULL << 16));
        for (int i = 0; i < medium_skips; i++) {
            seed = (seed * LCG_SKIP_2_16_MULTIPLIER + LCG_SKIP_2_16_ADDEND) & LCG_MASK;
        }
        dfz_initial_start -= large_skips * (1ULL << 16);

        // Handle the rest of the lcg skips (up to 2^16 -1 iterations here)
        for (int i = 0; i < dfz_initial_start; i++) {
            seed = (seed * LCG_MULTIPLIER + LCG_ADDEND) & LCG_MASK;
        }
    }

    boinc_end_critical_section();


    // Block group being a group of cuda blocks
    int block_groups_needed = (total_blocks + BLOCKS_PER_GROUP - 1) / BLOCKS_PER_GROUP;
    if (DEBUG) {
        fprintf(stderr, "Doing %d total block groups, starting at %d, with %d blocks each containing %d threads.\n", block_groups_needed, block_group_start, BLOCKS_PER_GROUP, THREADS_PER_BLOCK);
    }
    time_t start_time = time(NULL);
    // unsigned long long seed = 112962399045242;  // Polymetrics test seed

    for (int block_group = block_group_start; block_group < block_groups_needed; block_group++) {

        // Minimize the extra work we are doing (yay efficiency)
        int blocks_to_do = BLOCKS_PER_GROUP;
        if ((block_group + 1) * BLOCKS_PER_GROUP > total_blocks) {
            blocks_to_do = (block_group + 1) * BLOCKS_PER_GROUP - total_blocks;
        }

        if (DEBUG) {
            fprintf(stderr, "%d %d %llu %llu\n", blocks_to_do, THREADS_PER_BLOCK, THREAD_WORK, TOTAL_WORK);
            fprintf(stderr, "[Block %d]: Searching %llu seeds.\n", block_group + 1, blocks_to_do * THREADS_PER_BLOCK * THREAD_WORK);
        }

        // Launch kernel
        if (DEBUG) {
            fprintf(stderr, "Starting seed: %llu.\n", seed);
        }
        gpu_bruteforce<<<blocks_to_do, THREADS_PER_BLOCK>>>(seed, counter, seed_list, missing_count_list, lcg_skip_list_multiplier, lcg_skip_list_addend);
        hipDeviceSynchronize();

        boinc_begin_critical_section();


        time_t elapsed = time(NULL) - start_time;

        // Write block group to checkpoint
        fp = boinc_fopen("flower_checkpoint.txt", "w+");

        struct checkpoint_vars data_store;
        data_store.offset = block_group + 1;
        data_store.dfz_tracker = dfz_tracker;
        data_store.seed = seed;
        data_store.elapsed_chkpoint = elapsed_chkpoint + elapsed;
        data_store.total_counter = total_counter;

        fwrite(&data_store, sizeof(data_store), 1, fp);
        fclose(fp);

        boinc_checkpoint_completed();


        // Check outputs
        if (*counter > 0) {
            fp = boinc_fopen("output.txt", "a");
            total_counter += *counter;
            for (int i = 0; i < *counter; i++) {
                fprintf(fp, "Found seed. DFZ: %016llu, matches: %d.\n", dfz_tracker + seed_list[i], FLOWER_COUNT - missing_count_list[i]);
            }

            fclose(fp);
        }
        *counter = 0;

        // Boinc reporting
        boinc_fraction_done((double) block_group / block_groups_needed);

        boinc_end_critical_section();
    }

    

    boinc_begin_critical_section();

    // Calculate seeds per second (millions per second)
    time_t elapsed = time(NULL) - start_time;

    time_t total_time = elapsed_chkpoint + elapsed;

    double speed = (double) (TOTAL_WORK / 1000000) / total_time;
    
    printf("Found %llu valid seeds.\n", total_counter);
    printf("Finished in %llu seconds. Scanned %.2f million seeds per second.\n", (long long) total_time, speed);

    // Free memory
    hipFree(counter);
    hipFree(seed_list);

    // Delete checkpoint file
    boinc_delete_file("flower_checkpoint.txt");

    boinc_end_critical_section();
}



int main(int argc, char *argv[] ) {


    unsigned long long dfz = 0;
    int x = 0;
    int y = 0;
    int z = 0;
    int gpu_device;

    // Parse arguments
    for (int i = 1; i < argc; i += 2) {
        const char *param = argv[i];
        if (strcmp(param, "--dfz") == 0) {
            dfz = atoi(argv[i + 1]);
        } else if (strcmp(param, "-x") == 0) {
            x = atoi(argv[i + 1]);
        } else if (strcmp(param, "-y") == 0) {
            y = atoi(argv[i + 1]);
        } else if (strcmp(param, "-z") == 0) {
            z = atoi(argv[i + 1]);
        } else if (strcmp(param, "-d") == 0 || strcmp(param, "--device") == 0){
            gpu_device = atoi(argv[i + 1]);
        }
    }
	#ifdef BOINC
	APP_INIT_DATA aid;
	boinc_get_init_data(aid);
	
	if (aid.gpu_device_num >= 0) {
		gpu_device = aid.gpu_device_num;
		fprintf(stderr,"boinc gpu %i gpuindex: %i \n", aid.gpu_device_num, gpu_device);
		} else {
		fprintf(stderr,"stndalone gpuindex %i \n", gpu_device);
	}
	#endif
    hipSetDevice(gpu_device);
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipDeviceSynchronize());
    center[0] = x;
    center[1] = y;
    center[2] = z;

    fprintf(stderr, "Received arguments: dfz: %llu, center: %d, %d, %d.\n", dfz, x, y, z);


    // Do the gpu thing
    run_kernel(dfz);
    fprintf(stderr, "Finished task!\n");

    boinc_finish(0);
}